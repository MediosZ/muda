#include "hip/hip_runtime.h"
#include <numeric>
#include <catch2/catch.hpp>
#include <muda/muda.h>
#include <muda/container.h>
#include <muda/cub/cub.h>
#undef max
#undef min
using namespace muda;


struct Reducable
{
    int id = -1;
    int data;
};

bool operator==(const Reducable& lhs, const Reducable& rhs)
{
    return lhs.id == rhs.id;
}

void device_reduce_reduce(Reducable& h_output, Reducable& gt_output)
{
    device_buffer buffer;
    size_t        size = 100;

    host_vector<Reducable> gt_input(size);

    device_vector<Reducable> input;
    device_var<Reducable>    output;

    std::for_each(gt_input.begin(),
                  gt_input.end(),
                  [](Reducable& r) { r.id = std::rand(); });
    input = gt_input;

    on().next<DeviceReduce>()
        .Reduce(
            buffer,
            data(input),
            data(output),
            size,
            [] __device__(const Reducable& l, const Reducable& r) -> Reducable
            { return l.id > r.id ? l : r; },
            Reducable{})
        .wait();

    gt_output = *std::max_element(gt_input.begin(),
                                  gt_input.end(),
                                  [](const Reducable& l, const Reducable& r)
                                  { return l.id < r.id; });

    h_output = output;
}


void device_reduce_min(float& h_output, float& gt_output)
{
    device_buffer buffer;
    size_t        size = 100;

    host_vector<float> gt_input(size);

    device_vector<float> input;
    device_var<float>    output;

    std::for_each(
        gt_input.begin(), gt_input.end(), [](float& r) { r = std::rand(); });
    input = gt_input;

    on().next<DeviceReduce>().Min(buffer, data(input), data(output), size).wait();

    gt_output = *std::min_element(gt_input.begin(), gt_input.end());

    h_output = output;
}

void device_reduce_max(float& h_output, float& gt_output)
{
    device_buffer buffer;
    size_t        size = 100;

    host_vector<float> gt_input(size);

    device_vector<float> input;
    device_var<float>    output;

    std::for_each(
        gt_input.begin(), gt_input.end(), [](float& r) { r = std::rand(); });
    input = gt_input;

    on().next<DeviceReduce>().Max(buffer, data(input), data(output), size).wait();

    gt_output = *std::max_element(gt_input.begin(), gt_input.end());

    h_output = output;
}


void device_reduce_sum(float& h_output, float& gt_output)
{
    device_buffer buffer;
    size_t        size = 100;

    host_vector<float> gt_input(size);

    device_vector<float> input;
    device_var<float>    output;

    std::for_each(
        gt_input.begin(), gt_input.end(), [](float& r) { r = std::rand(); });
    input = gt_input;

    on().next<DeviceReduce>().Sum(buffer, data(input), data(output), size).wait();

    gt_output = std::accumulate(gt_input.begin(), gt_input.end(), 0.0f);

    h_output = output;
}

void device_reduce_argmin(int& h_output, int& gt_output)
{
    using KVP = hipcub::KeyValuePair<int, float>;
    device_buffer buffer;
    size_t        size = 100;

    host_vector<float>   gt_input(size);
    device_vector<float> input;
    device_var<KVP>      output;
    KVP                  h_output_kvp;


    std::for_each(
        gt_input.begin(), gt_input.end(), [](float& r) { r = std::rand(); });
    input = gt_input;

    // using std to get the index of the min element
    gt_output = std::min_element(gt_input.begin(), gt_input.end()) - gt_input.begin();


    on().next<DeviceReduce>().ArgMin(buffer, data(input), data(output), size).wait();

    h_output_kvp = output;
    h_output     = h_output_kvp.key;
}

void device_reduce_argmax(int& h_output, int& gt_output)
{
    using KVP = hipcub::KeyValuePair<int, float>;
    device_buffer buffer;
    size_t        size = 100;

    host_vector<float>   gt_input(size);
    device_vector<float> input;
    device_var<KVP>      output;
    KVP                  h_output_kvp;

    std::for_each(
        gt_input.begin(), gt_input.end(), [](float& r) { r = std::rand(); });
    input = gt_input;

    // using std to get the index of the max element
    gt_output = std::max_element(gt_input.begin(), gt_input.end()) - gt_input.begin();


    on().next<DeviceReduce>().ArgMax(buffer, data(input), data(output), size).wait();

    h_output_kvp = output;
    h_output     = h_output_kvp.key;
}

// CustomMin functor
struct CustomMin
{
    template <typename T>
    __host__ __device__ __forceinline__ T operator()(const T& a, const T& b) const
    {
        return (b < a) ? b : a;
    }
};

void device_reduce_reduce_by_key(host_vector<int>& h_unique_out,
                                 host_vector<int>& h_aggregates_out,
                                 int&              h_num_runs_out,
                                 host_vector<int>& gt_unique_out,
                                 host_vector<int>& gt_aggregates_out,
                                 int&              gt_num_runs_out)
{
    device_buffer buffer;
    size_t        size = 100;

    host_vector<float>   gt_input(size);
    device_vector<float> input;

    // Declare, allocate, and initialize device-accessible pointers for input and output
    int num_items = 8;  // e.g., 8

    std::vector<int>   keys_in          = {0, 2, 2, 9, 5, 5, 5, 8};
    std::vector<int>   values_in        = {0, 7, 1, 6, 2, 5, 3, 4};
    device_vector<int> d_keys_in        = keys_in;
    device_vector<int> d_values_in      = values_in;
    device_vector<int> d_unique_out     = keys_in;
    device_vector<int> d_aggregates_out = keys_in;
    device_var<int>    d_num_runs_out;
    CustomMin          reduction_op;

    on().next<DeviceReduce>()
        .ReduceByKey(buffer,
                     data(d_keys_in),
                     data(d_unique_out),
                     data(d_values_in),
                     data(d_aggregates_out),
                     data(d_num_runs_out),
                     reduction_op,
                     num_items)
        .wait();

    d_unique_out.resize(d_num_runs_out);
    d_aggregates_out.resize(d_num_runs_out);

    h_unique_out     = d_unique_out;
    h_aggregates_out = d_aggregates_out;
    h_num_runs_out   = d_num_runs_out;

    gt_unique_out     = std::vector<int>{0, 2, 9, 5, 8};
    gt_aggregates_out = std::vector<int>{0, 1, 6, 2, 4};
    gt_num_runs_out   = 5;
}

TEST_CASE("device_reduce", "[cub]")
{
    SECTION("Reduce")
    {
        Reducable h_output;
        Reducable gt_output;
        device_reduce_reduce(h_output, gt_output);
        REQUIRE(h_output.id == gt_output.id);
    }

    SECTION("Min")
    {
        float h_output;
        float gt_output;
        device_reduce_min(h_output, gt_output);
        REQUIRE(h_output == gt_output);
    }

    SECTION("Max")
    {
        float h_output;
        float gt_output;
        device_reduce_max(h_output, gt_output);
        REQUIRE(h_output == gt_output);
    }

    SECTION("Sum")
    {
        float h_output;
        float gt_output;
        device_reduce_sum(h_output, gt_output);
        REQUIRE(h_output == gt_output);
    }

    SECTION("ArgMin")
    {
        int h_output;
        int gt_output;
        device_reduce_argmin(h_output, gt_output);
        REQUIRE(h_output == gt_output);
    }

    SECTION("ArgMax")
    {
        int h_output;
        int gt_output;
        device_reduce_argmax(h_output, gt_output);
        REQUIRE(h_output == gt_output);
    }

    SECTION("ReduceByKey")
    {
        host_vector<int> h_unique_out;
        host_vector<int> h_aggregates_out;
        int              h_num_runs_out;
        host_vector<int> gt_unique_out;
        host_vector<int> gt_aggregates_out;
        int              gt_num_runs_out;
        device_reduce_reduce_by_key(
            h_unique_out, h_aggregates_out, h_num_runs_out, gt_unique_out, gt_aggregates_out, gt_num_runs_out);
        REQUIRE(h_unique_out == gt_unique_out);
        REQUIRE(h_aggregates_out == gt_aggregates_out);
        REQUIRE(h_num_runs_out == gt_num_runs_out);
    }
}

void device_scan_inclusive_sum(host_vector<float>& h_output, host_vector<float>& gt_output)
{
    device_buffer buffer;
    size_t        size = 100;

    host_vector<float> gt_input(size);
    gt_output.resize(size);
    device_vector<float> input(size);
    device_vector<float> output(size);

    std::for_each(
        gt_input.begin(), gt_input.end(), [](float& r) { r = std::rand(); });
    input = gt_input;

    // using std to get the inclusive sum
    std::partial_sum(gt_input.begin(), gt_input.end(), gt_output.begin());

    on().next<DeviceScan>()
        .InclusiveSum(buffer, data(input), data(output), size)
        .wait();

    h_output = output;
}

void device_scan_exclusive_sum(host_vector<float>& h_output, host_vector<float>& gt_output)
{
    device_buffer buffer;
    size_t        size = 100;

    host_vector<float> gt_input(size);
    gt_output.resize(size);
    device_vector<float> input(size);
    device_vector<float> output(size);

    std::for_each(
        gt_input.begin(), gt_input.end(), [](float& r) { r = std::rand(); });
    input = gt_input;

    // using std to get the exclusive sum
    gt_output[0] = 0;
    std::partial_sum(gt_input.begin(), gt_input.end() - 1, gt_output.begin() + 1);

    on().next<DeviceScan>()
        .ExclusiveSum(buffer, data(input), data(output), size)
        .wait();

    h_output = output;
}

void device_scan_inclusive_scan(host_vector<float>& h_output, host_vector<float>& gt_output)
{
    device_buffer buffer;
    size_t        size = 100;

    host_vector<float> gt_input(size);
    gt_output.resize(size);
    device_vector<float> input(size);
    device_vector<float> output(size);

    std::for_each(
        gt_input.begin(), gt_input.end(), [](float& r) { r = std::rand(); });
    input = gt_input;

    // using std to get the inclusive scan
    std::partial_sum(gt_input.begin(), gt_input.end(), gt_output.begin());

    on().next<DeviceScan>()
        .InclusiveScan(
            buffer,
            data(input),
            data(output),
            [] __device__(const float& a, const float& b) { return a + b; },
            size)
        .wait();

    h_output = output;
}

void device_scan_exclusive_scan(host_vector<float>& h_output, host_vector<float>& gt_output)
{
    device_buffer buffer;
    size_t        size = 100;

    host_vector<float> gt_input(size);
    gt_output.resize(size);
    device_vector<float> input(size);
    device_vector<float> output(size);

    std::for_each(
        gt_input.begin(), gt_input.end(), [](float& r) { r = std::rand(); });
    input = gt_input;

    // using std to get the exclusive scan
    gt_output[0] = 0;
    std::partial_sum(gt_input.begin(), gt_input.end() - 1, gt_output.begin() + 1);

    on().next<DeviceScan>()
        .ExclusiveScan(
            buffer,
            data(input),
            data(output),
            [] __device__(const float& a, const float& b) { return a + b; },
            0.0f,
            size)
        .wait();

    h_output = output;
}


void device_scan_exclusive_sum_by_key(host_vector<int>& h_values_out,
                                      host_vector<int>& gt_values_out)
{
    device_buffer buffer;
    size_t        size = 8;

    host_vector<int> h_keys_in   = std::vector{0, 2, 2, 9, 5, 5, 5, 8};
    host_vector<int> h_values_in = std::vector{0, 7, 1, 6, 2, 5, 3, 4};
    gt_values_out                = std::vector{0, 0, 7, 0, 0, 2, 7, 0};

    device_vector<int> d_keys_in   = h_keys_in;
    device_vector<int> d_values_in = h_values_in;
    device_vector<int> d_keys_out(size);
    device_vector<int> d_values_out(size);

    on().next<DeviceScan>()
        .ExclusiveSumByKey(buffer, data(d_keys_in), data(d_values_in), data(d_values_out), size)
        .wait();
    h_values_out = d_values_out;
}


void device_scan_inclusive_sum_by_key(host_vector<int>& h_values_out,
                                      host_vector<int>& gt_values_out)
{
    device_buffer buffer;
    size_t        size = 8;

    host_vector<int> h_keys_in   = std::vector{0, 2, 2, 9, 5, 5, 5, 8};
    host_vector<int> h_values_in = std::vector{0, 7, 1, 6, 2, 5, 3, 4};
    gt_values_out                = std::vector{0, 7, 8, 6, 2, 7, 10, 4};

    device_vector<int> d_keys_in   = h_keys_in;
    device_vector<int> d_values_in = h_values_in;
    device_vector<int> d_keys_out(size);
    device_vector<int> d_values_out(size);

    on().next<DeviceScan>()
        .InclusiveSumByKey(buffer, data(d_keys_in), data(d_values_in), data(d_values_out), size)
        .wait();

    h_values_out = d_values_out;
}


void device_scan_exclusive_scan_by_key(host_vector<int>& h_values_out,
                                       host_vector<int>& gt_values_out)
{
    device_buffer buffer;
    size_t        size = 8;

    host_vector<int> h_keys_in   = std::vector{0, 2, 2, 9, 5, 5, 5, 8};
    host_vector<int> h_values_in = std::vector{0, 7, 1, 6, 2, 5, 3, 4};
    gt_values_out                = std::vector{0, 0, 7, 0, 0, 2, 7, 0};

    device_vector<int> d_keys_in   = h_keys_in;
    device_vector<int> d_values_in = h_values_in;
    device_vector<int> d_keys_out(size);
    device_vector<int> d_values_out(size);

    on().next<DeviceScan>()
        .ExclusiveScanByKey(
            buffer,
            data(d_keys_in),
            data(d_values_in),
            data(d_values_out),
            [] __device__(const int& a, const int& b) -> int { return a + b; },
            0,
            size)
        .wait();

    h_values_out = d_values_out;
}


void device_scan_inclusive_scan_by_key(host_vector<int>& h_values_out,
                                       host_vector<int>& gt_values_out)
{
    device_buffer buffer;
    size_t        size = 8;

    host_vector<int> h_keys_in   = std::vector{0, 2, 2, 9, 5, 5, 5, 8};
    host_vector<int> h_values_in = std::vector{0, 7, 1, 6, 2, 5, 3, 4};
    gt_values_out                = std::vector{0, 7, 8, 6, 2, 7, 10, 4};

    device_vector<int> d_keys_in   = h_keys_in;
    device_vector<int> d_values_in = h_values_in;
    device_vector<int> d_keys_out(size);
    device_vector<int> d_values_out(size);


    on().next<DeviceScan>()
        .InclusiveScanByKey(
            buffer,
            data(d_keys_in),
            data(d_values_in),
            data(d_values_out),
            [] __device__(const int& a, const int& b) { return a + b; },
            size)
        .wait();

    h_values_out = d_values_out;
}

TEST_CASE("device_scan", "[cub]")
{
    SECTION("InclusiveSum")
    {
        host_vector<float> h_output, gt_output;
        device_scan_inclusive_sum(h_output, gt_output);
        REQUIRE(h_output == gt_output);
    }

    SECTION("ExclusiveSum")
    {
        host_vector<float> h_output, gt_output;
        device_scan_exclusive_sum(h_output, gt_output);
        REQUIRE(h_output == gt_output);
    }


    SECTION("InclusiveScan")
    {
        host_vector<float> h_output, gt_output;
        device_scan_inclusive_scan(h_output, gt_output);
        REQUIRE(h_output == gt_output);
    }


    SECTION("ExclusiveScan")
    {
        host_vector<float> h_output, gt_output;
        device_scan_exclusive_scan(h_output, gt_output);
        REQUIRE(h_output == gt_output);
    }

    SECTION("ExclusiveSumByKey")
    {
        host_vector<int> h_values_out, gt_values_out;
        device_scan_exclusive_sum_by_key(h_values_out, gt_values_out);
        REQUIRE(h_values_out == gt_values_out);
    }

    SECTION("InclusiveSumByKey")
    {
        host_vector<int> h_values_out, gt_values_out;
        device_scan_inclusive_sum_by_key(h_values_out, gt_values_out);
        REQUIRE(h_values_out == gt_values_out);
    }

    SECTION("ExclusiveScanByKey")
    {
        host_vector<int> h_values_out, gt_values_out;
        device_scan_exclusive_scan_by_key(h_values_out, gt_values_out);
        REQUIRE(h_values_out == gt_values_out);
    }

    SECTION("InclusiveScanByKey")
    {
        host_vector<int> h_values_out, gt_values_out;
        device_scan_inclusive_scan_by_key(h_values_out, gt_values_out);
        REQUIRE(h_values_out == gt_values_out);
    }
}


void device_run_length_encode_encode(host_vector<int>& h_unique_out,
                                     host_vector<int>& h_counts_out,
                                     int&              h_num_runs_out,
                                     host_vector<int>& gt_unique_out,
                                     host_vector<int>& gt_counts_out,
                                     int&              gt_num_runs_out)
{
    device_buffer buffer;
    size_t        size = 8;

    host_vector<int> h_input = std::vector{0, 2, 2, 9, 5, 5, 5, 8};
    gt_unique_out            = std::vector{0, 2, 9, 5, 8};
    gt_counts_out            = std::vector{1, 2, 1, 3, 1};
    gt_num_runs_out          = 5;

    device_vector<int> d_input = h_input;
    device_vector<int> d_unique_out(size);
    device_vector<int> d_counts_out(size);
    device_var<int>    d_num_runs_out;

    on().next<DeviceRunLengthEncode>()
        .Encode(buffer, data(d_input), data(d_unique_out), data(d_counts_out), data(d_num_runs_out), size)
        .wait();

    d_unique_out.resize(d_num_runs_out);
    d_counts_out.resize(d_num_runs_out);

    h_unique_out   = d_unique_out;
    h_counts_out   = d_counts_out;
    h_num_runs_out = d_num_runs_out;
}

void device_run_length_encode_non_trivial_runs(host_vector<int>& h_offsets_out,
                                               host_vector<int>& h_counts_out,
                                               int&              h_num_runs_out,
                                               host_vector<int>& gt_offsets_out,
                                               host_vector<int>& gt_counts_out,
                                               int& gt_num_runs_out)
{
    device_buffer buffer;
    size_t        size = 8;

    host_vector<int> h_input = std::vector{0, 2, 2, 9, 5, 5, 5, 8};
    gt_offsets_out           = std::vector{1, 4};
    gt_counts_out            = std::vector{2, 3};
    gt_num_runs_out          = 2;

    device_vector<int> d_input = h_input;
    device_vector<int> d_offsets_out(size);
    device_vector<int> d_counts_out(size);
    device_var<int>    d_num_runs_out;

    on().next<DeviceRunLengthEncode>()
        .NonTrivialRuns(
            buffer, data(d_input), data(d_offsets_out), data(d_counts_out), data(d_num_runs_out), size)
        .wait();

    d_offsets_out.resize(d_num_runs_out);
    d_counts_out.resize(d_num_runs_out);

    h_offsets_out  = d_offsets_out;
    h_counts_out   = d_counts_out;
    h_num_runs_out = d_num_runs_out;
}

TEST_CASE("device_run_length_encode", "[cub]")
{
    SECTION("Encode")
    {
        host_vector<int> h_unique_out, h_counts_out, gt_unique_out, gt_counts_out;
        int h_num_runs_out, gt_num_runs_out;
        device_run_length_encode_encode(
            h_unique_out, h_counts_out, h_num_runs_out, gt_unique_out, gt_counts_out, gt_num_runs_out);
        REQUIRE(h_unique_out == gt_unique_out);
        REQUIRE(h_counts_out == gt_counts_out);
        REQUIRE(h_num_runs_out == gt_num_runs_out);
    }

    SECTION("NonTrivialRuns")
    {
        host_vector<int> h_offsets_out, h_counts_out, gt_offsets_out, gt_counts_out;
        int h_num_runs_out, gt_num_runs_out;
        device_run_length_encode_non_trivial_runs(
            h_offsets_out, h_counts_out, h_num_runs_out, gt_offsets_out, gt_counts_out, gt_num_runs_out);
        REQUIRE(h_offsets_out == gt_offsets_out);
        REQUIRE(h_counts_out == gt_counts_out);
        REQUIRE(h_num_runs_out == gt_num_runs_out);
    }
}

void device_radix_sort_sort_pairs(host_vector<int>&   h_keys_out,
                                  host_vector<float>& h_values_out,
                                  host_vector<int>&   gt_keys_out,
                                  host_vector<float>& gt_values_out)
{
    size_t size = 100;

    // Generate random input data
    host_vector<int>   h_keys_in(size);
    host_vector<float> h_values_in(size);
    std::for_each(
        h_keys_in.begin(), h_keys_in.end(), [](int& r) { r = std::rand(); });
    std::for_each(h_values_in.begin(),
                  h_values_in.end(),
                  [](float& r) { r = std::rand(); });

    // Sort input data using std::sort
    gt_keys_out   = h_keys_in;
    gt_values_out = h_values_in;
    std::vector<size_t> indices(size);
    std::iota(indices.begin(), indices.end(), 0);
    std::sort(indices.begin(),
              indices.end(),
              [&](size_t a, size_t b) { return h_keys_in[a] < h_keys_in[b]; });
    for(size_t i = 0; i < size; ++i)
    {
        gt_keys_out[i]   = h_keys_in[indices[i]];
        gt_values_out[i] = h_values_in[indices[i]];
    }

    // Sort input data using DeviceRadixSort::SortPairs
    device_vector<int>   d_keys_in   = h_keys_in;
    device_vector<float> d_values_in = h_values_in;
    device_vector<int>   d_keys_out(size);
    device_vector<float> d_values_out(size);
    device_buffer        buffer;

    on().next<DeviceRadixSort>()
        .SortPairs(buffer, data(d_keys_in), data(d_keys_out), data(d_values_in), data(d_values_out), size)
        .wait();

    // Copy results from device to host
    h_keys_out   = d_keys_out;
    h_values_out = d_values_out;
}


void device_radix_sort_sort_pairs_descending(host_vector<int>&   h_keys_out,
                                             host_vector<float>& h_values_out,
                                             host_vector<int>&   gt_keys_out,
                                             host_vector<float>& gt_values_out)
{
    size_t size = 100;

    // Generate random input data
    host_vector<int>   h_keys_in(size);
    host_vector<float> h_values_in(size);
    std::for_each(
        h_keys_in.begin(), h_keys_in.end(), [](int& r) { r = std::rand(); });
    std::for_each(h_values_in.begin(),
                  h_values_in.end(),
                  [](float& r) { r = std::rand(); });

    // Sort input data using std::sort in descending order
    gt_keys_out   = h_keys_in;
    gt_values_out = h_values_in;
    std::vector<size_t> indices(size);
    std::iota(indices.begin(), indices.end(), 0);
    std::sort(indices.begin(),
              indices.end(),
              [&](size_t a, size_t b) { return h_keys_in[a] > h_keys_in[b]; });
    for(size_t i = 0; i < size; ++i)
    {
        gt_keys_out[i]   = h_keys_in[indices[i]];
        gt_values_out[i] = h_values_in[indices[i]];
    }

    // Sort input data using DeviceRadixSort::SortPairsDescending
    device_vector<int>   d_keys_in   = h_keys_in;
    device_vector<float> d_values_in = h_values_in;
    device_vector<int>   d_keys_out(size);
    device_vector<float> d_values_out(size);
    device_buffer        buffer;

    on().next<DeviceRadixSort>()
        .SortPairsDescending(
            buffer, data(d_keys_in), data(d_keys_out), data(d_values_in), data(d_values_out), size)
        .wait();

    // Copy results from device to host
    h_keys_out   = d_keys_out;
    h_values_out = d_values_out;
}


void device_radix_sort_sort_keys(host_vector<int>& h_keys_out, host_vector<int>& gt_keys_out)
{
    size_t size = 100;

    // Generate random input data
    host_vector<int> h_keys_in(size);
    std::for_each(
        h_keys_in.begin(), h_keys_in.end(), [](int& r) { r = std::rand(); });

    // Sort input data using std::sort
    gt_keys_out = h_keys_in;
    std::sort(gt_keys_out.begin(), gt_keys_out.end());

    // Sort input data using DeviceRadixSort::SortKeys
    device_vector<int> d_keys_in = h_keys_in;
    device_vector<int> d_keys_out(size);
    device_buffer      buffer;

    on().next<DeviceRadixSort>()
        .SortKeys(buffer, data(d_keys_in), data(d_keys_out), size)
        .wait();

    // Copy results from device to host
    h_keys_out = d_keys_out;
}


void device_radix_sort_sort_keys_descending(host_vector<int>& h_keys_out,
                                            host_vector<int>& gt_keys_out)
{
    size_t size = 100;

    // Generate random input data
    host_vector<int> h_keys_in(size);
    std::for_each(
        h_keys_in.begin(), h_keys_in.end(), [](int& r) { r = std::rand(); });

    // Sort input data using std::sort in descending order
    gt_keys_out = h_keys_in;
    std::sort(gt_keys_out.begin(), gt_keys_out.end(), std::greater<int>());

    // Sort input data using DeviceRadixSort::SortKeysDescending
    device_vector<int> d_keys_in = h_keys_in;
    device_vector<int> d_keys_out(size);
    device_buffer      buffer;

    on().next<DeviceRadixSort>()
        .SortKeysDescending(buffer, data(d_keys_in), data(d_keys_out), size)
        .wait();

    // Copy results from device to host
    h_keys_out = d_keys_out;
}

TEST_CASE("device_radix_sort", "[cub]")
{

    SECTION("SortPairsDescending")
    {
        host_vector<int>   h_keys_out;
        host_vector<float> h_values_out;
        host_vector<int>   gt_keys_out;
        host_vector<float> gt_values_out;

        device_radix_sort_sort_pairs_descending(h_keys_out, h_values_out, gt_keys_out, gt_values_out);

        REQUIRE(h_keys_out == gt_keys_out);
        REQUIRE(h_values_out == gt_values_out);
    }

    SECTION("SortPairs")
    {
        host_vector<int>   h_keys_out;
        host_vector<float> h_values_out;
        host_vector<int>   gt_keys_out;
        host_vector<float> gt_values_out;

        device_radix_sort_sort_pairs(h_keys_out, h_values_out, gt_keys_out, gt_values_out);
        // Check if the results are equal
        REQUIRE(h_keys_out == gt_keys_out);
        REQUIRE(h_values_out == gt_values_out);
    }

    SECTION("SortKeys")
    {
        host_vector<int> h_keys_out, gt_keys_out;
        device_radix_sort_sort_keys(h_keys_out, gt_keys_out);
        REQUIRE(h_keys_out == gt_keys_out);
    }

    SECTION("SortKeysDescending")
    {
        host_vector<int> h_keys_out, gt_keys_out;
        device_radix_sort_sort_keys_descending(h_keys_out, gt_keys_out);
        REQUIRE(h_keys_out == gt_keys_out);
    }
}


void device_merge_sort_sort_pairs(host_vector<int>&   h_keys_out,
                                  host_vector<float>& h_values_out,
                                  host_vector<int>&   gt_keys_out,
                                  host_vector<float>& gt_values_out)
{
    size_t size = 100;

    // Generate random input data
    host_vector<int>   h_keys_in(size);
    host_vector<float> h_values_in(size);
    std::for_each(
        h_keys_in.begin(), h_keys_in.end(), [](int& r) { r = std::rand(); });
    std::for_each(h_values_in.begin(),
                  h_values_in.end(),
                  [](float& r) { r = std::rand(); });

    // Sort input data using std::sort
    gt_keys_out   = h_keys_in;
    gt_values_out = h_values_in;
    std::vector<size_t> indices(size);
    std::iota(indices.begin(), indices.end(), 0);
    std::sort(indices.begin(),
              indices.end(),
              [&](size_t a, size_t b) { return h_keys_in[a] < h_keys_in[b]; });
    for(size_t i = 0; i < size; ++i)
    {
        gt_keys_out[i]   = h_keys_in[indices[i]];
        gt_values_out[i] = h_values_in[indices[i]];
    }

    // Sort input data using DeviceMergeSort::SortPairs
    device_vector<int>   d_keys   = h_keys_in;
    device_vector<float> d_values = h_values_in;
    device_buffer        buffer;

    on().next<DeviceMergeSort>()
        .SortPairs(buffer,
                   data(d_keys),
                   data(d_values),
                   size,
                   [] __device__(auto l, auto r) { return l < r; })
        .wait();

    // Copy results from device to host
    h_keys_out   = d_keys;
    h_values_out = d_values;
}


void device_merge_sort_sort_pairs_copy(host_vector<int>&   h_keys_out,
                                       host_vector<float>& h_values_out,
                                       host_vector<int>&   gt_keys_out,
                                       host_vector<float>& gt_values_out)
{
    size_t size = 100;

    // Generate random input data
    host_vector<int>   h_keys_in(size);
    host_vector<float> h_values_in(size);
    std::for_each(
        h_keys_in.begin(), h_keys_in.end(), [](int& r) { r = std::rand(); });
    std::for_each(h_values_in.begin(),
                  h_values_in.end(),
                  [](float& r) { r = std::rand(); });

    // Sort input data using std::sort
    gt_keys_out   = h_keys_in;
    gt_values_out = h_values_in;
    std::vector<size_t> indices(size);
    std::iota(indices.begin(), indices.end(), 0);
    std::sort(indices.begin(),
              indices.end(),
              [&](size_t a, size_t b) { return h_keys_in[a] < h_keys_in[b]; });
    for(size_t i = 0; i < size; ++i)
    {
        gt_keys_out[i]   = h_keys_in[indices[i]];
        gt_values_out[i] = h_values_in[indices[i]];
    }

    // Sort input data using DeviceMergeSort::SortPairsCopy
    device_vector<int>   d_keys_in   = h_keys_in;
    device_vector<float> d_values_in = h_values_in;
    device_vector<int>   d_keys_out(size);
    device_vector<float> d_values_out(size);
    device_buffer        buffer;

    on().next<DeviceMergeSort>()
        .SortPairsCopy(buffer,
                       data(d_keys_in),
                       data(d_values_in),
                       data(d_keys_out),
                       data(d_values_out),
                       size,
                       [] __device__(auto l, auto r) { return l < r; })
        .wait();

    // Copy results from device to host
    h_keys_out   = d_keys_out;
    h_values_out = d_values_out;
}

void device_merge_sort_sort_keys(host_vector<int>& h_keys_out, host_vector<int>& gt_keys_out)
{
    size_t size = 100;

    // Generate random input data
    host_vector<int> h_keys_in(size);
    std::for_each(
        h_keys_in.begin(), h_keys_in.end(), [](int& r) { r = std::rand(); });

    // Sort input data using std::sort
    gt_keys_out = h_keys_in;
    std::sort(gt_keys_out.begin(), gt_keys_out.end());

    // Sort input data using DeviceMergeSort::SortKeys
    device_vector<int> d_keys = h_keys_in;
    device_buffer      buffer;

    on().next<DeviceMergeSort>()
        .SortKeys(buffer,
                  data(d_keys),
                  size,
                  [] __device__(auto l, auto r) { return l < r; })
        .wait();

    // Copy results from device to host
    h_keys_out = d_keys;
}

void device_merge_sort_sort_keys_copy(host_vector<int>& h_keys_out, host_vector<int>& gt_keys_out)
{
    size_t size = 100;

    // Generate random input data
    host_vector<int> h_keys_in(size);
    std::for_each(
        h_keys_in.begin(), h_keys_in.end(), [](int& r) { r = std::rand(); });

    // Sort input data using std::sort
    gt_keys_out = h_keys_in;
    std::sort(gt_keys_out.begin(), gt_keys_out.end());

    // Sort input data using DeviceMergeSort::SortKeysCopy
    device_vector<int> d_keys_in = h_keys_in;
    device_vector<int> d_keys_out(size);
    device_buffer      buffer;

    on().next<DeviceMergeSort>()
        .SortKeysCopy(buffer,
                      data(d_keys_in),
                      data(d_keys_out),
                      size,
                      [] __device__(auto l, auto r) { return l < r; })
        .wait();

    // Copy results from device to host
    h_keys_out = d_keys_out;
}

void device_merge_sort_stable_sort_pairs(host_vector<int>&   h_keys_out,
                                         host_vector<float>& h_values_out,
                                         host_vector<int>&   gt_keys_out,
                                         host_vector<float>& gt_values_out)
{
    size_t size = 100;

    // Generate random input data
    host_vector<int>   h_keys_in(size);
    host_vector<float> h_values_in(size);
    std::for_each(
        h_keys_in.begin(), h_keys_in.end(), [](int& r) { r = std::rand(); });
    std::for_each(h_values_in.begin(),
                  h_values_in.end(),
                  [](float& r) { r = std::rand(); });

    // Sort input data using std::stable_sort
    gt_keys_out   = h_keys_in;
    gt_values_out = h_values_in;
    std::vector<size_t> indices(size);
    std::iota(indices.begin(), indices.end(), 0);
    std::stable_sort(indices.begin(),
                     indices.end(),
                     [&](size_t a, size_t b)
                     { return h_keys_in[a] < h_keys_in[b]; });
    for(size_t i = 0; i < size; ++i)
    {
        gt_keys_out[i]   = h_keys_in[indices[i]];
        gt_values_out[i] = h_values_in[indices[i]];
    }

    // Sort input data using DeviceMergeSort::StableSortPairs
    device_vector<int>   d_keys   = h_keys_in;
    device_vector<float> d_values = h_values_in;
    device_buffer        buffer;

    on().next<DeviceMergeSort>()
        .StableSortPairs(buffer,
                         data(d_keys),
                         data(d_values),
                         size,
                         [] __device__(auto l, auto r) { return l < r; })
        .wait();

    // Copy results from device to host
    h_keys_out   = d_keys;
    h_values_out = d_values;
}

void device_merge_sort_stable_sort_keys(host_vector<int>& h_keys_out,
                                        host_vector<int>& gt_keys_out)
{
    size_t size = 100;

    // Generate random input data
    host_vector<int> h_keys_in(size);
    std::for_each(
        h_keys_in.begin(), h_keys_in.end(), [](int& r) { r = std::rand(); });

    // Sort input data using std::stable_sort
    gt_keys_out = h_keys_in;
    std::stable_sort(gt_keys_out.begin(), gt_keys_out.end());

    // Sort input data using DeviceMergeSort::StableSortKeys
    device_vector<int> d_keys = h_keys_in;
    device_buffer      buffer;

    on().next<DeviceMergeSort>()
        .StableSortKeys(buffer,
                        data(d_keys),
                        size,
                        [] __device__(auto l, auto r) { return l < r; })
        .wait();

    // Copy results from device to host
    h_keys_out = d_keys;
}

TEST_CASE("device_merge_sort", "[cub]")
{
    SECTION("SortPairs")
    {
        host_vector<int>   h_keys_out;
        host_vector<float> h_values_out;
        host_vector<int>   gt_keys_out;
        host_vector<float> gt_values_out;

        device_merge_sort_sort_pairs(h_keys_out, h_values_out, gt_keys_out, gt_values_out);

        REQUIRE(h_keys_out == gt_keys_out);
        REQUIRE(h_values_out == gt_values_out);
    }

    SECTION("SortPairsCopy")
    {
        host_vector<int>   h_keys_out;
        host_vector<float> h_values_out;
        host_vector<int>   gt_keys_out;
        host_vector<float> gt_values_out;

        device_merge_sort_sort_pairs_copy(h_keys_out, h_values_out, gt_keys_out, gt_values_out);

        REQUIRE(h_keys_out == gt_keys_out);
        REQUIRE(h_values_out == gt_values_out);
    }

    SECTION("SortKeys")
    {
        host_vector<int> h_keys_out;
        host_vector<int> gt_keys_out;

        device_merge_sort_sort_keys(h_keys_out, gt_keys_out);

        REQUIRE(h_keys_out == gt_keys_out);
    }

    SECTION("SortKeysCopy")
    {
        host_vector<int> h_keys_out, gt_keys_out;
        device_merge_sort_sort_keys_copy(h_keys_out, gt_keys_out);
        REQUIRE(h_keys_out == gt_keys_out);
    }

    SECTION("StableSortPairs")
    {
        host_vector<int>   h_keys_out;
        host_vector<float> h_values_out;
        host_vector<int>   gt_keys_out;
        host_vector<float> gt_values_out;

        device_merge_sort_stable_sort_pairs(h_keys_out, h_values_out, gt_keys_out, gt_values_out);

        REQUIRE(h_keys_out == gt_keys_out);
        REQUIRE(h_values_out == gt_values_out);
    }

    SECTION("StableSortKeys")
    {
        host_vector<int> h_keys_out, gt_keys_out;
        device_merge_sort_stable_sort_keys(h_keys_out, gt_keys_out);
        REQUIRE(h_keys_out == gt_keys_out);
    }
}

void device_select_flagged(host_vector<int>& h_keys_out, host_vector<int>& gt_keys_out)
{
    size_t size = 100;

    // Generate random input data
    host_vector<int> h_keys_in(size);
    std::for_each(
        h_keys_in.begin(), h_keys_in.end(), [](int& r) { r = std::rand(); });

    // Generate flags
    host_vector<bool> h_flags(size);
    std::for_each(
        h_flags.begin(), h_flags.end(), [](bool& r) { r = std::rand() % 2; });

    // Filter input data using std::copy_if
    gt_keys_out.reserve(size);
    size_t idx = 0;
    std::copy_if(h_keys_in.begin(),
                 h_keys_in.end(),
                 std::back_inserter(gt_keys_out),
                 [&](auto key) { return h_flags[idx++]; });

    // Filter input data using DeviceSelect::Flagged
    device_vector<int>  d_keys_in = h_keys_in;
    device_vector<bool> d_flags   = h_flags;
    device_vector<int>  d_keys_out(size);
    device_var<int>     d_num_selected_out;
    device_buffer       buffer;

    on().next<DeviceSelect>()
        .Flagged(buffer, data(d_keys_in), data(d_flags), data(d_keys_out), data(d_num_selected_out), size)
        .wait();

    d_keys_out.resize(d_num_selected_out);
    // Copy results from device to host
    h_keys_out = d_keys_out;
}


void device_select_if(host_vector<int>& h_keys_out, host_vector<int>& gt_keys_out)
{
    size_t size = 100;

    // Generate random input data
    host_vector<int> h_keys_in(size);
    std::for_each(
        h_keys_in.begin(), h_keys_in.end(), [](int& r) { r = std::rand(); });

    // Filter input data using std::copy_if
    gt_keys_out.reserve(size);
    std::copy_if(h_keys_in.begin(),
                 h_keys_in.end(),
                 std::back_inserter(gt_keys_out),
                 [](int key) { return key % 2 == 0; });

    // Filter input data using DeviceSelect::If
    device_vector<int> d_keys_in = h_keys_in;
    device_vector<int> d_keys_out(size);
    device_var<int>    d_num_selected_out;
    device_buffer      buffer;

    on().next<DeviceSelect>()
        .If(buffer,
            data(d_keys_in),
            data(d_keys_out),
            data(d_num_selected_out),
            size,
            [] __device__(auto key) { return key % 2 == 0; })
        .wait();

    d_keys_out.resize(d_num_selected_out);
    // Copy results from device to host
    h_keys_out = d_keys_out;
}

void device_select_unique(host_vector<int>& h_keys_out, host_vector<int>& gt_keys_out)
{
    size_t size = 100;

    // Generate random input data
    host_vector<int> h_keys_in(size);
    std::for_each(
        h_keys_in.begin(), h_keys_in.end(), [](int& r) { r = std::rand(); });

    // Filter input data using std::unique
    gt_keys_out.reserve(size);
    std::unique_copy(h_keys_in.begin(), h_keys_in.end(), std::back_inserter(gt_keys_out));

    // Filter input data using DeviceSelect::Unique
    device_vector<int> d_keys_in = h_keys_in;
    device_vector<int> d_keys_out(size);
    device_var<int>    d_num_selected_out;
    device_buffer      buffer;

    on().next<DeviceSelect>()
        .Unique(buffer, data(d_keys_in), data(d_keys_out), data(d_num_selected_out), size)
        .wait();

    d_keys_out.resize(d_num_selected_out);
    // Copy results from device to host
    h_keys_out = d_keys_out;
}


TEST_CASE("device_select", "[cub]")
{
    SECTION("Flagged")
    {
        host_vector<int> h_keys_out, gt_keys_out;
        device_select_flagged(h_keys_out, gt_keys_out);
        REQUIRE(h_keys_out == gt_keys_out);
    }

    SECTION("If")
    {
        host_vector<int> h_keys_out, gt_keys_out;
        device_select_if(h_keys_out, gt_keys_out);
        REQUIRE(h_keys_out == gt_keys_out);
    }

    SECTION("Unique")
    {
        host_vector<int> h_keys_out, gt_keys_out;
        device_select_unique(h_keys_out, gt_keys_out);
        REQUIRE(h_keys_out == gt_keys_out);
    }
}

void device_partition_if(host_vector<int>& h_keys_out, host_vector<int>& gt_keys_out)
{
    size_t size = 100;

    // Generate random input data
    host_vector<int> h_keys_in(size);
    std::for_each(
        h_keys_in.begin(), h_keys_in.end(), [](int& r) { r = std::rand(); });

    // Partition input data using std::partition
    gt_keys_out   = h_keys_in;
    size_t select = 0;
    std::partition(gt_keys_out.begin(),
                   gt_keys_out.end(),
                   [&](int key)
                   {
                       if(key % 2 == 0)
                       {
                           ++select;
                           return true;
                       }
                       return false;
                   });
    gt_keys_out.resize(select);
    std::sort(gt_keys_out.begin(), gt_keys_out.end());

    // Partition input data using DevicePartition::If
    device_vector<int> d_keys_in = h_keys_in;
    device_vector<int> d_keys_out(size);
    device_var<int>    d_num_selected_out;
    device_buffer      buffer;

    on().next<DevicePartition>()
        .If(buffer,
            data(d_keys_in),
            data(d_keys_out),
            data(d_num_selected_out),
            size,
            [] __device__(auto key) { return key % 2 == 0; })
        .wait();

    d_keys_out.resize(d_num_selected_out);
    // Copy results from device to host
    h_keys_out = d_keys_out;
    std::sort(h_keys_out.begin(), h_keys_out.end());
}

TEST_CASE("device_partition", "[cub]")
{
    SECTION("If")
    {
        host_vector<int> h_keys_out, gt_keys_out;
        device_partition_if(h_keys_out, gt_keys_out);
        REQUIRE(h_keys_out == gt_keys_out);
    }
}
